#include "hip/hip_runtime.h"
#include "pathtracer.cuh" // very important first include for cuda to work

namespace pathtracer {
	// constants
	__shared__ float3* accum_buffer_dev;
	__shared__ uint8_t* display_buffer_dev;
	__shared__ uint8_t* pathtracer_buffer_dev;

	Triangle* cudaTriangleList;
	int* cudaTrianglesIndex;
	BVH_Node* cudaBVHNodes;
	Material* cudaMaterialList;

	__device__ int RootandUsedNodes_dev[2];

	// host functions
	void initCuda(kernelParams& params) {
		printf("Init cuda...\n");

		// perf metrics
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start);

		// device
		hipError_t error = hipSetDevice(0);
		if (error != hipSuccess) {
			printf("ERROR : no device compatible !!");
		}
		// memory alloc
		hipMalloc((void**)&accum_buffer_dev, params.windowSize.x * params.windowSize.y * sizeof(float3));
		hipMalloc((void**)&display_buffer_dev, params.windowSize.x * params.windowSize.y * 4 * sizeof(uint8_t));
		hipMalloc((void**)&pathtracer_buffer_dev, params.windowSize.x * params.windowSize.y * 4 * sizeof(uint8_t));

		hipEventRecord(stop);
		hipEventSynchronize(stop);

		float miliS = 0;
		hipEventElapsedTime(&miliS, start, stop);

		printf("Initialisation done in %f ms !\nInformations : \n", miliS);
		printf("    Ray size : %i bytes,\n    Material size : %i bytes,\n    Hit size : %i bytes.\n", sizeof(Ray), sizeof(Material), sizeof(Hit));
	}
	void transferTriangles(int* trianglesI, Triangle* allTri, int nbTri) {
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start);

		hipMalloc((void**)&cudaTriangleList, nbTri * sizeof(Triangle));
		hipMemcpy(cudaTriangleList, allTri, nbTri * sizeof(Triangle), hipMemcpyHostToDevice);

		hipMalloc((void**)&cudaTrianglesIndex, nbTri * sizeof(int));
		hipMemcpy(cudaTrianglesIndex, trianglesI, nbTri * sizeof(int), hipMemcpyHostToDevice);
		
		hipEventRecord(stop);

		hipEventSynchronize(stop);

		float ms = 0;
		hipEventElapsedTime(&ms, start, stop);
		printf("Triangles tranfered in %f ms \n", ms);
	}
	void tranfertMaterials(Material* hostMatList, int nbMat) {
		hipMalloc((void**)&cudaMaterialList, nbMat * sizeof(Material));
		hipMemcpy(cudaMaterialList, hostMatList, nbMat * sizeof(Material), hipMemcpyHostToDevice);
		printf("Materials transfered !\n");
	}
	void tranfertBVH(BVH_Node* nodes_host, int rootNIdx_host, int nodesUsed_host, int nbTri) {
		//hipMemcpyToSymbol(HIP_SYMBOL(Nodes_dev), nodes_host, (N * 2 - 1) * sizeof(BVH_Node));
		int p[2] = { rootNIdx_host, nodesUsed_host };
		hipMemcpyToSymbol(HIP_SYMBOL(RootandUsedNodes_dev), p, 2 * sizeof(int));

		hipMalloc((void**)&cudaBVHNodes, (nbTri * 2 - 1) * sizeof(BVH_Node));
		hipMemcpy(cudaBVHNodes, nodes_host, (nbTri * 2 - 1) * sizeof(BVH_Node), hipMemcpyHostToDevice);
		printf("BVH transfered !!\n");
	}
	
	void endCuda() {

		hipError_t error = hipFree(accum_buffer_dev);
		if (error != hipSuccess) printf("CAN'T FREE ACCUM_BUFFER_DEV IN pathtracer.cu : %s", hipGetErrorString(error));
		error = hipFree(display_buffer_dev);
		if (error != hipSuccess) printf("CAN'T FREE DISPLAY_BUFFER_DEV IN pathtracer.cu : %s", hipGetErrorString(error));
		error = hipFree(pathtracer_buffer_dev);
		if (error != hipSuccess) printf("CAN'T FREE PATHTRACER_BUFFER_DEV IN pathtracer.cu : %s", hipGetErrorString(error));

		hipFree(cudaTriangleList);
		hipFree(cudaBVHNodes);
		hipFree(cudaTrianglesIndex);
		hipFree(cudaMaterialList);

		printf("Everything has been free !\n");
	}
	// device code

	// device constants (like triangles or something)
	

	//pathtracer

	__device__ inline bool triangleIntersect(Hit& hit, Ray ray, Triangle tri, Material mat) {
		float3 edge1 = tri.b - tri.a;
		float3 edge2 = tri.c - tri.a;
		float3 h = cross(ray.d, edge2);
		float a = dot(edge1, h);
		if (a > -0.0001f && a < 0.0001f) return false;
		float f = 1. / a;
		float3 s = ray.o - tri.a;
		float u = f * dot(s, h);
		if (u < 0. || u > 1) return false;
		float3 q = cross(s, edge1);
		float v = f * dot(ray.d, q);
		if (v < 0 || u + v > 1) return false;
		float t = f * dot(edge2, q);

		if (t < 0. || t > hit.t) return false;
		float3 oldAbsorption = hit.mat.extinction;

		hit.t = t;
		hit.mat = mat;
		hit.hit = true;

		float w = 1. - u - v;

		if (mat.useTexture) {
			float2 samplePos = tri.tA * w + tri.tB * u + tri.tC * v;
			float3 pos = ray.o + ray.d * t;
			float4 read = tex2D<float4>(mat.diffuseTexture, samplePos.x, samplePos.y);
			hit.mat.baseColor = make_float3(read.x, read.y, read.z);
			//hit.mat.baseColor = make_float3(samplePos.x, samplePos.y, 0.);
		}

		//some shit to refine here
		float3 normalCalc = normalize(cross(edge1, edge2));
		float mult = -sign(dot(ray.d, normalCalc));
		normalCalc *= mult;
		float3 normalFile = normalize(tri.nA * w + tri.nB * u + tri.nC * v) * mult;
		hit.normal = dot(normalFile, ray.d) < 0.f ? normalFile : normalCalc;
		//hit.normal = normalCalc;

		return true;

	}

	__device__ inline bool BVHIntersect(Hit& hit, Ray ray, int3& debug, Triangle* cudaTriList, int* cudaTriIndex, BVH_Node* cudaNodes, Material* cudaMats) {
		/*
		BVH_Node* node = &Nodes_dev[0], *stack[N];
		// slow asf
		//BVH_Node** stack = (BVH_Node**)malloc(64);
		int stackPtr = 0;


		// DONT WORK FUCKKKKK
		if (boxIntersectF(hit.t, ray, node->aabbMin, node->aabbMax) < 1e30) {
			while (1) {
				//if (boxIntersectF(hit.t, ray, node->aabbMin, node->aabbMax) != 1e30) {
				debug.x++;
				if (node->triCount >= 1) {
					for (int i = 0; i < node->triCount; i++) {
						if (triangleIntersect(hit, ray, dev_TRI[dev_ABC[i + node->firstTriIdx]], mat)) {
							debug.y++;
						}
					}

					if (stackPtr == 0) break; else node = stack[--stackPtr];
					continue;
				}
				BVH_Node* child1 = &Nodes_dev[node->leftNode];
				BVH_Node* child2 = &Nodes_dev[node->leftNode + 1];
				float dist1 = boxIntersectF(hit.t, ray, child1->aabbMin, child1->aabbMax);
				float dist2 = boxIntersectF(hit.t, ray, child2->aabbMin, child2->aabbMax);
				if (dist1 > dist2) {
					float d = dist1; dist1 = dist2; dist2 = d;
					BVH_Node* c = child1; child1 = child2; child2 = c;
				}
				if (dist1 == 1e30) {
					if (stackPtr == 0) break;
					else node = stack[--stackPtr];
				}
				else {

					node = child1;
					if (dist2 != 1e30) {
						stack[stackPtr++] = child2;
					}
				}
			}
		}
		*/


		//BVH_Node stack[10];
		int stack[10];
		int stackIdx = 0;
		stack[stackIdx++] = 0;

		float dst = hit.t;
		float3 normal = make_float3(0., 0., 0.);
		bool hasIntersected = false;
		float2 minMax = make_float2(-1, dst);

		float3 invDir = 1. / ray.d;

		while (stackIdx > 0)
		{
			BVH_Node node = cudaNodes[stack[--stackIdx]];
			if (boxIntersectF(hit.t, ray, node.aabbMin, node.aabbMax, invDir) < hit.t) {
				if (node.triCount > 0) { // leaf node
					for (int i = 0; i < node.triCount; i++) { // leaf node
						triangleIntersect(hit, ray, cudaTriList[cudaTriIndex[i + node.leftFirst]], cudaMats[cudaTriList[cudaTriIndex[i + node.leftFirst]].matIndex]);
						/*float4 curr = triangleIntersect_noMat(ray, dev_TRI[dev_ABC[i + node.firstTriIdx]]);

						if(curr.w > 0.) {
							minMax = make_float2(fmaxf(minMax.x, dst), fminf(minMax.y, dst));
							if (curr.w < dst) {
								dst = curr.w;
								normal = make_float3(curr.x, curr.y, curr.z);
								hasIntersected = true;
							}
						}*/
					}
					debug.z++;
				}
				else {
					//stack[stackIdx++] = Nodes_dev[node.leftNode];
					//stack[stackIdx++] = Nodes_dev[node.leftNode + 1];

					BVH_Node childLeft = cudaNodes[node.leftFirst];
					BVH_Node childRight = cudaNodes[node.leftFirst + 1];

					float dstLeft = boxIntersectF(hit.t, ray, childLeft.aabbMin, childLeft.aabbMax, invDir);
					float dstRight = boxIntersectF(hit.t, ray, childRight.aabbMin, childRight.aabbMax, invDir);
					int left = node.leftFirst, right = node.leftFirst + 1;

					/*if (dstLeft > dstRight) {
						int c = right, right = left, left = c;
						float cf = dstRight, dstRight = dstLeft, dstLeft = cf;
					}
					if (dstLeft < hit.t) stack[stackIdx++] = left;
					if (dstRight < hit.t) stack[stackIdx++] = right;*/
					if (dstLeft > dstRight) {
						if (dstLeft < hit.t) stack[stackIdx++] = left;
						if (dstRight < hit.t) stack[stackIdx++] = right;
					}
					else {
						if (dstRight < hit.t) stack[stackIdx++] = right;
						if (dstLeft < hit.t) stack[stackIdx++] = left;
					}

				}
				debug.x++;
			}
		}

		/*if (hasIntersected) {
			bool inside = minMax.x <= minMax.y;
			hit.normal = normal;
			hit.t = dst;

			hit.mat = mat;
			hit.mat.absorption = inside ? mat.absorption : make_float3(0., 0., 0.);
			//hit.mat.absorption = mat.absorption;
			hit.mat.n = inside ? hit.mat.IOR : 1. / hit.mat.IOR;
			hit.hit = true;
		}*/

	}
	__device__ inline Hit map(Ray ray, int3 &debug, Triangle* cudaTriList, int* cudaTriIndex, BVH_Node* cudaNodes, Material* cudaMats) {
		Hit hit;
		//hit.mat = Material(make_float3(1., 1., 1.), make_float3(1., 1., 1.), 0.0f, 0.0f, 0.0f, make_float3(0., 0., 0.), 0.0f, 1.0f, make_float3(0., 0., 0.), make_float3(1.5f, 1.5f, 1.5f));
		hit.mat = Material();
		
		/*sphereIntersect(hit, ray, make_float4(-4., 0.5, 0., 0.5), Material(make_float3(0.2, 0.8, 0.4), make_float3(1., 1., 1.), 0.4, 1., 1., make_float3(0., 0., 0.), 0., 1., make_float3(0., 0., 0.), 1.5));
		sphereIntersect(hit, ray, make_float4(-2.5, 0.5, 0., 0.5), Material(make_float3(0.2, 0.8, 0.4), make_float3(1., 1., 1.), 0.4, 1., 0.95, make_float3(0., 0., 0.), 0., 1., make_float3(0., 0., 0.), 1.5));
		sphereIntersect(hit, ray, make_float4(-1., 0.5, 0., 0.5), Material(make_float3(0.2, 0.8, 0.4), make_float3(1., 1., 1.), 0.4, 1., 0.9, make_float3(0., 0., 0.), 0., 1., make_float3(0., 0., 0.), 1.5));
		sphereIntersect(hit, ray, make_float4(0.5, 0.5, 0., 0.5), Material(make_float3(0.2, 0.8, 0.4), make_float3(1., 1., 1.), 0.4, 1., 0.85, make_float3(0., 0., 0.), 0., 1., make_float3(0., 0., 0.), 1.5));
		sphereIntersect(hit, ray, make_float4(2., 0.5, 0., 0.5), Material(make_float3(0.2, 0.8, 0.4), make_float3(1., 1., 1.), 0.4, 1., 0.8, make_float3(0., 0., 0.), 0., 1., make_float3(0., 0., 0.), 1.5));
		sphereIntersect(hit, ray, make_float4(3.5, 0.5, 0., 0.5), Material(make_float3(0.2, 0.8, 0.4), make_float3(1., 1., 1.), 0.4, 1., 0.75, make_float3(0., 0., 0.), 0., 1., make_float3(0., 0., 0.), 1.5));


		*/
		//triangleIntersect(hit, ray, Triangle(make_float3(0., 0., 0.), make_float3(3., 0., 0.), make_float3(0., 0., 3.)), Material(make_float3(0.2, 0.3, 0.7), make_float3(1., 1., 1.), 0.02, 0.6, 0.98, make_float3(0., 0., 0.), 0., 1., make_float3(0., 0., 0.), 1.5));

		//boxIntersect(hit, ray, make_float3(-100.0f, 270.0f, -100.0f), make_float3(100.0f, 290.f, 100.0f), Material(make_float3(1., 1., 1.), make_float3(1., 1., 1.), 0.0, 0., 0., make_float3(0.f, 0.f, 0.f), 0.f, 1.f, make_float3(15.f, 15.f, 15.f), make_float3(1.5f, 1.5f, 1.5f)));

		//boxIntersect(hit, ray, make_float3(-400.f, 100.f, 0.f), make_float3(-270.f, 200.f, 230.f), Material(make_float3(1., 1., 1.), make_float3(1., 1., 1.), 0.0, 0., 0., make_float3(0.f, 0.f, 0.f), 0.f, 1.f, make_float3(15.f, 15.f, 15.f), make_float3(1.5f, 1.5f, 1.5f)));
		//boxIntersect(hit, ray, make_float3(-100.0f, 270.0f, -100.0f), make_float3(100.0f, 290.f, 100.0f), Material(make_float3(1., 1., 1.), make_float3(1., 1., 1.), 0.0, 0., 0., make_float3(0.f, 0.f, 0.f), 0.f, 1.f, make_float3(15.f, 15.f, 15.f), make_float3(1.5f, 1.5f, 1.5f)));
		//sphereIntersect(hit, ray, make_float4(18.3, 8., 11.7, 0.75), Material(make_float3(1., 1., 1.), make_float3(1., 1., 1.), 0., 0., 0.9, make_float3(0., 0., 0.), 0., 1., make_float3(10., 2.8, 0.039) * 2., 1.5));
		//sphereIntersect(hit, ray, make_float4(2.6, 15.1, 20.3, 0.75), Material(make_float3(1., 1., 1.), make_float3(1., 1., 1.), 0., 0., 0.9, make_float3(0., 0., 0.), 0., 1., make_float3(10., 2.8, 0.039) * 2., 1.5));
		
		//planeIntersect(hit, ray, make_float3(0., 1., 0.), -2., Material(make_float3(0.1, 0.1, 0.2), make_float3(1., 1., 1.), 0.0, 0., 0., make_float3(0.f, 0.f, 0.f), 0.f, 1.f, make_float3(0.f, 0.f, 0.f), make_float3(1.5f, 1.5f, 1.5f)));
		BVHIntersect(hit, ray, debug, cudaTriList, cudaTriIndex, cudaNodes, cudaMats);
		//__syncthreads();

		/*Material sphMat;
		sphMat.metallic = 0.;
		sphMat.baseColor = make_float3(1., 0.4, 0.6);
		sphMat.roughness = 0.4;
		sphMat.anisotropic = 0.;
		sphereIntersect(hit, ray, make_float4(-400., 50., 0., 50.), sphMat);
		sphMat.anisotropic = 0.2;
		sphereIntersect(hit, ray, make_float4(-250., 50., 0., 50.), sphMat);
		sphMat.anisotropic = 0.4;
		sphereIntersect(hit, ray, make_float4(-100., 50., 0., 50.), sphMat);
		sphMat.anisotropic = 0.6;
		sphereIntersect(hit, ray, make_float4(50., 50., 0., 50.), sphMat);
		sphMat.anisotropic = 0.8;
		sphereIntersect(hit, ray, make_float4(200., 50., 0., 50.), sphMat);
		sphMat.anisotropic = 0.99;
		sphereIntersect(hit, ray, make_float4(350., 50., 0., 50.), sphMat);
		
		Material boxMat;
		boxMat.emissive = make_float3(10.f, 10.f, 10.f);
		boxMat.baseColor = make_float3(1.f, 1.f, 1.f);
		boxIntersect(hit, ray, make_float3(-100.0f, 270.0f, -100.0f), make_float3(100.0f, 290.f, 100.0f), boxMat);*/
		
		return hit;
	}
	__device__ float HG(float g, float sundotrd) {
		float gg = g * g;	return (1. - gg) / pow(1. + gg - 2. * g * sundotrd, 1.5);
	}
	__device__ float rayleigh(float u) {
		return 0.75 * (1. + clamp(u * u, 0., 1.));
	}
	__device__ float3 skyGradient(float3 rd, kernelParams params) {
		float3 sunColor = mix(make_float3(1., 0.9, 0.8), make_float3(1., 0.2, 0.), exp(-abs(params.sunDirection.y) * 5.));
		float costh0 = dot(rd, params.sunDirection);
		float blend = smoothstep(-1., 0.6, costh0);
		float mult = smoothstep(-0.5, 0.7, costh0) * 0.5 + 0.5;
		float3 day = mix(make_float3(0.3, 0.6, 1.) * 0.7 * rayleigh(costh0), make_float3(0.9, 0.95, 1.), exp(-abs(rd.y) * 5.));
		float3 even = mix(make_float3(0.3, 0.6, 1.) * 0.7, mix(make_float3(0.2, 0.3, 0.4), make_float3(1., 0.2, 0.), blend), exp(-abs(rd.y) * 5.));
		float3 sun = 0.02 / length(rd - params.sunDirection) * sunColor ;

		float3 night = make_float3(0.05, 0.1, 0.2) + clamp(0.02 / length(rd + params.sunDirection), 0., 1.) * make_float3(0.7, 0.8, 1.) * 0.6;
		float3 dayF = mix(day, even, exp(-abs(params.sunDirection.y) * 5.)) + sun;

		float earth = smoothstep(-0.02, 0.02, rd.y) * 0.5 + 0.5;

		float mieM = 1. - abs(dot(params.sunDirection, make_float3(0., 1., 0.))) + 0.3;
		return mix(night, dayF + HG(0.6, costh0) * sunColor * 0.3 * mieM, smoothstep(-0.2, 0.0, params.sunDirection.y)) * earth * 0.75;
		//return vec3(HG(0.9,costh0)) ;
	}
	__device__ inline float3 skyColor(float3 rayDir, kernelParams params) {
		return make_float3(0.f, 0.f, 0.f);
		//float3 skyGrad = mix(make_float3(0.4, 0.7, 1.), make_float3(0.8, 0.9, 1.), exp(-abs(rayDir.y) * 5.)) * 0.5;
		//return skyGrad * 3.;
		//float sineV = fmax(sin(rayDir.x * 7.) * sin(rayDir.z * 7.), 0.);
		//float sineV = fmaxf(sin(rayDir.x * 18.), 0.f);
		//return make_float3(sineV, sineV, sineV);
		//return make_float3(1., 1., 1.) * fmax((double)rayDir.y, 0.);
		//return dot(rayDir, params.sunDirection) > 0.95 ? make_float3(20., 20., 20.) : make_float3(0.,0.,0.);
		//return skyGrad + smoothstep(1., 0.9, length(rayDir - params.sunDirection) / 0.25) * 50.;
		//return skyGradient(rayDir, params);
		//return make_float3(5.f, 5.f, 5.f);
	}
	__device__ inline void pathtrace(float& result, kernelParams params, Ray ray, Rand_state& state, int channel, Triangle* cudaTriList, int* cudaTriIndex, BVH_Node* cudaNodes, Material* cudaMats) {
		float rayColor = 1.0;

		Hit info;
		bool inside = false;
		int3 nullVal = make_int3(0, 0, 0);
		for (int i = 0; i < 5; i++) {
			info = map(ray, nullVal, cudaTriList, cudaTriIndex, cudaNodes, cudaMats);
			info.isInside = inside;

			if (!info.hit) {
				result += getN(skyColor(ray.d, params), channel) * rayColor;
				return;
			}

			ray.o += ray.d * info.t;
			info.mat.roughness = fmaxf(info.mat.roughness, 0.001f);

			if (info.isInside)
				rayColor *= exp(-getN(info.mat.extinction * info.t, channel));
			/*
			// material prop
			float specularChance = 0.;
			float refractionChance = info.mat.refractionProba;

			float rayProba = 1.;
			if (info.mat.f90 > 0.) {
				specularChance = FresnelReflectAmount(info.isInside ? getN(info.mat.IOR, channel) : 1.0f, 
					!info.isInside ? getN(info.mat.IOR, channel) : 1.0f, 
					ray, info);
			}
			//float isSpecular = rand(&state) < specularChance ? 1. : 0.;
			float isSpecular = 0.;
			float isRefractive = 0.;
			float zeta = randC(&state);
			if (specularChance > 0. && zeta < specularChance) {
				isSpecular = 1.;
				rayProba = specularChance;
				ray.o += info.normal * 0.1;
			}
			else if (refractionChance > 0. && zeta < refractionChance + specularChance) {
				isRefractive = 1.;
				rayProba = refractionChance;
				ray.o -= info.normal * 0.1;
			}
			else {
				rayProba = 1. - (specularChance + refractionChance);
				ray.o += info.normal * 0.1;
			}

			float3 diffuseRay = normalize(info.normal + generateUniformSample(state));
			float3 specularRay = reflect(ray.d, info.normal);
			specularRay = normalize(mix(diffuseRay, specularRay, info.mat.specularSmoothness * info.mat.specularSmoothness));

			float3 refractionRayDir = refract(ray.d, info.normal, getN(info.isInside ? info.mat.IOR : 1. / info.mat.IOR, channel));

			ray.d = mix(diffuseRay, specularRay, isSpecular);
			if (dot(refractionRayDir, refractionRayDir) == 0.f) refractionRayDir = ray.d; // should be useless unless f90 == 0
			ray.d = mix(ray.d, refractionRayDir, isRefractive);

			result += getN(info.mat.emissive, channel) * rayColor;

			if (isRefractive < 0.5)
				rayColor *= mix(getN(info.mat.albedo, channel), getN(info.mat.specularAlbedo, channel), isSpecular);
			else inside = !inside;

			rayProba = fmax(rayProba, 0.0001f);
			rayColor /= rayProba;
			*/


			float3 L;
			float pdf;
			float3 bsdf = Disney::DisneySample(info, ray.d * -1.0, info.normal, L, pdf, state, ray.o);
			inside = info.isInside;

			ray.d = L;

			result += getN(info.mat.emissive, channel) * rayColor;

			if (pdf > 0.f) rayColor *= getN(bsdf / pdf, channel);
			else return;

			ray.o += ray.d * 0.1f;

			// russian roulette 
			{
				float p = rayColor;
				if (randC(&state) > p)
					return;

				rayColor *= 1.0f / p;
			}
		}
		return;
	}
	__device__ inline float3 pixelColor(kernelParams params, Ray ray, Rand_state& state, Triangle* cudaTriList, int* cudaTriIndex, BVH_Node* cudaNodes, Material* cudaMats) {
		float3 color = make_float3(0., 0., 0.);
		if (!params.isRendering) {
			int3 debugV = make_int3(0, 0, 0);
			Hit info = map(ray, debugV, cudaTriList, cudaTriIndex, cudaNodes, cudaMats);
			color = info.normal * 0.5 + 0.5;
			//float4 test = tex2D<float4>(cudaMats[0].diffuseTexture, 0.f, 0.f);
			//color = dot(ray.d, info.normal) < 0.f ? make_float3(0.f, 0.f, 1.f) : make_float3(1.f, 0.f, 0.f);
			//if (dot(info.normal, info.normal) == 0.) color = skyGradient(ray.d, params);
			//if(info.t > params.focalDistance)color = mix(color, make_float3(0.5, 1., 0.5), 0.7);
			//color.x = info.mat.absorption.x;
			//color.y = clamp(info.t / 50., 0.0f, 1.0f);
			//color = info.mat.albedo;
			//color.x = clamp(info.t / 5., 0., 1.);
			//color = make_float3((float)debugV.x / 10.0f, (float)debugV.y / 10.0f, (float)debugV.z);
		}
		else {
			int channel = int(randC(&state) * 3.);
			switch (channel)
			{
			case 0:
				pathtrace(color.x, params, ray, state, 0, cudaTriList, cudaTriIndex, cudaNodes, cudaMats);
				break;
			case 1:
				pathtrace(color.y, params, ray, state, 1, cudaTriList, cudaTriIndex, cudaNodes, cudaMats);
				break;
			case 2:
				pathtrace(color.z, params, ray, state, 2, cudaTriList, cudaTriIndex, cudaNodes, cudaMats);
				break;
			}
			//pathtrace(color.x, params, ray, state, 0);
			//pathtrace(color.y, params, ray, state, 1);
			//pathtrace(color.z, params, ray, state, 2);
			color *= 3.;
		}

		return color;
	}
	__global__ inline void renderPixel(kernelParams params, uint8_t* ptBuffer, float3* accumBuff, Triangle* cudaTriList, int* cudaTriIndex, BVH_Node* cudaNodes, Material* cudaMats) {
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;
		//__syncthread();
		if (x > params.windowSize.x && y > params.windowSize.y) return;

		int idx = x + y * params.windowSize.x;

		Rand_state rand_state;
		hiprand_init(idx, 0, 4096 * params.frameIndex, &rand_state);

		float2 uv = make_float2((float)x / params.windowSize.x, 1. - (float)y / params.windowSize.y);
		float2 uvCam = uv - 0.5;
		uvCam.x *= (float)params.windowSize.x / params.windowSize.y;

		// rayDirection computations
		float fov = 1.;
		float3 cameraTarget = params.rayOrigin + make_float3(sin(params.cameraAngle.x) * cos(params.cameraAngle.y), sin(params.cameraAngle.y), -cos(params.cameraAngle.x) * cos(params.cameraAngle.y));
		float3 ww = normalize(cameraTarget - params.rayOrigin);
		float3 uu = normalize(cross(ww, make_float3(0., 1., 0.)));
		float3 vv = normalize(cross(uu, ww));
		float3 rayDirection = normalize(uu * uvCam.x + vv * uvCam.y + ww * fov);

		/*if (params.isRendering) {
			float3 target = params.rayOrigin + rayDirection * params.focalDistance;
			params.rayOrigin += generateUniformSample(rand_state) * params.DOF_strenght;
			rayDirection = normalize(target - params.rayOrigin);
		}*/
		float3 color0 = max3(pixelColor(params, Ray(params.rayOrigin, rayDirection), rand_state, cudaTriList, cudaTriIndex, cudaNodes, cudaMats), make_float3(0.f,0.f,0.f));

		__syncthreads();
		if (params.frameIndex == 0 || !params.isRendering)
			accumBuff[idx] = color0;
		else
				accumBuff[idx] += color0;

		float3 color = accumBuff[idx] / (params.isRendering ? params.frameIndex + 1 : 1.);
		aces(color);
		// drawing to texture
		int colorX = color.x * 255;
		int colorY = color.y * 255;
		int colorZ = color.z * 255;

		ptBuffer[idx * 4] = uint8_t(clamp(colorX, 0, 255));
		ptBuffer[idx * 4 + 1] = uint8_t(clamp(colorY, 0, 255));
		ptBuffer[idx * 4 + 2] = uint8_t(clamp(colorZ, 0, 255));
		ptBuffer[idx * 4 + 3] = uint8_t(255);
	}
	

	__global__ inline void antialias(kernelParams params, uint8_t* ptBuffer, uint8_t* dispBuffer) {
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;
		//__syncthread();
		if (x > params.windowSize.x && y > params.windowSize.y) return;
		if (x <= 1 || x >= params.windowSize.x - 1 || y <= 1 || y >= params.windowSize.y - 1) return;

		int idx = x + y * params.windowSize.x;
		int left = idx - 1;
		int right = idx + 1;
		int top = idx - params.windowSize.x;
		int bottom = idx + params.windowSize.x;

		int colorR = (int)ptBuffer[idx * 4] +
					 (int)ptBuffer[left * 4] * 0.5 +
					 (int)ptBuffer[right * 4] * 0.5 +
					 (int)ptBuffer[top * 4] * 0.5 +
					 (int)ptBuffer[bottom * 4] * 0.5;

		int colorG = (int)ptBuffer[idx * 4 + 1] +
					 (int)ptBuffer[left * 4 + 1] * 0.5 +
					 (int)ptBuffer[right * 4 + 1] * 0.5 +
					 (int)ptBuffer[top * 4 + 1] * 0.5 +
					 (int)ptBuffer[bottom * 4 + 1] * 0.5;

		int colorB = (int)ptBuffer[idx * 4 + 2] +
					 (int)ptBuffer[left * 4 + 2] * 0.5 +
					 (int)ptBuffer[right * 4 + 2] * 0.5 +
					 (int)ptBuffer[top * 4 + 2] * 0.5 +
					 (int)ptBuffer[bottom * 4 + 2] * 0.5;

		dispBuffer[idx * 4] = uint8_t(colorR * 0.3333f);
		dispBuffer[idx * 4 + 1] = uint8_t(colorG * 0.3333f);
		dispBuffer[idx * 4 + 2] = uint8_t(colorB * 0.3333f);
		dispBuffer[idx * 4 + 3] = uint8_t(255);
	}
	void render(kernelParams params) {
		const int threadSize = 16;
		dim3 blockSize(8, 8, 1U);
		dim3 gridSize(int(params.windowSize.x / blockSize.x), int(params.windowSize.y / blockSize.y), 1U);
		renderPixel<<<gridSize, blockSize >>>(params, display_buffer_dev, accum_buffer_dev, cudaTriangleList, cudaTrianglesIndex, cudaBVHNodes, cudaMaterialList);
		
		//antialias<<<gridSize, blockSize>>>(params, pathtracer_buffer_dev, display_buffer_dev);
		if (hipPeekAtLastError() != hipSuccess) {
			printf("Error with kernel : %s \n", hipGetErrorString(hipGetLastError()));
		}

		hipDeviceSynchronize();
		// copy device display buff. to host
		hipError_t error = hipMemcpy(params.pixelBuffer, display_buffer_dev, params.windowSize.x * params.windowSize.y * 4 * sizeof(uint8_t), hipMemcpyDeviceToHost);
		if (error != hipSuccess) {
			printf("ERROR WHILE TRANSFERING DEVICE DATA TO HOST : %s \n", hipGetErrorString(error));
		}
	}
}