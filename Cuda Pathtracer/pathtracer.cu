#include "hip/hip_runtime.h"
#include "pathtracer.cuh" // very important first include for cuda to work

namespace pathtracer {
	// constants
	__shared__ float3* accum_buffer_dev;
	__shared__ uint8_t* display_buffer_dev;
	__shared__ uint8_t* pathtracer_buffer_dev;

	Triangle* cudaTriangleList;
	int* cudaTrianglesIndex;
	BVH_Node* cudaBVHNodes;
	Material* cudaMaterialList;

	__device__ int RootandUsedNodes_dev[2];

	// host functions
	void initCuda(kernelParams& params) {
		printf("Init cuda...\n");

		// perf metrics
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start);

		// device
		hipError_t error = hipSetDevice(0);
		if (error != hipSuccess) {
			printf("ERROR : no device compatible !!");
		}
		// memory alloc
		hipMalloc((void**)&accum_buffer_dev, params.windowSize.x * params.windowSize.y * sizeof(float3));
		hipMalloc((void**)&display_buffer_dev, params.windowSize.x * params.windowSize.y * 4 * sizeof(uint8_t));
		hipMalloc((void**)&pathtracer_buffer_dev, params.windowSize.x * params.windowSize.y * 4 * sizeof(uint8_t));

		hipEventRecord(stop);
		hipEventSynchronize(stop);

		float miliS = 0;
		hipEventElapsedTime(&miliS, start, stop);

		printf("Initialisation done in %f ms !\nInformations : \n", miliS);
		printf("    Ray size : %i bytes,\n    Material size : %i bytes,\n    Hit size : %i bytes.\n", sizeof(Ray), sizeof(Material), sizeof(Hit));
	}
	void transferTriangles(int* trianglesI, Triangle* allTri, int nbTri) {
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start);

		hipMalloc((void**)&cudaTriangleList, nbTri * sizeof(Triangle));
		hipMemcpy(cudaTriangleList, allTri, nbTri * sizeof(Triangle), hipMemcpyHostToDevice);

		hipMalloc((void**)&cudaTrianglesIndex, nbTri * sizeof(int));
		hipMemcpy(cudaTrianglesIndex, trianglesI, nbTri * sizeof(int), hipMemcpyHostToDevice);
		
		hipEventRecord(stop);

		hipEventSynchronize(stop);

		float ms = 0;
		hipEventElapsedTime(&ms, start, stop);
		printf("Triangles tranfered in %f ms \n", ms);
	}
	void tranfertMaterials(Material* hostMatList, int nbMat) {
		hipMalloc((void**)&cudaMaterialList, nbMat * sizeof(Material));
		hipMemcpy(cudaMaterialList, hostMatList, nbMat * sizeof(Material), hipMemcpyHostToDevice);
		printf("Materials transfered !\n");
	}
	void tranfertBVH(BVH_Node* nodes_host, int rootNIdx_host, int nodesUsed_host, int nbTri) {
		//hipMemcpyToSymbol(HIP_SYMBOL(Nodes_dev), nodes_host, (N * 2 - 1) * sizeof(BVH_Node));
		int p[2] = { rootNIdx_host, nodesUsed_host };
		hipMemcpyToSymbol(HIP_SYMBOL(RootandUsedNodes_dev), p, 2 * sizeof(int));

		hipMalloc((void**)&cudaBVHNodes, (nbTri * 2 - 1) * sizeof(BVH_Node));
		hipMemcpy(cudaBVHNodes, nodes_host, (nbTri * 2 - 1) * sizeof(BVH_Node), hipMemcpyHostToDevice);
		printf("BVH transfered !!\n");
	}
	
	void endCuda() {

		hipError_t error = hipFree(accum_buffer_dev);
		if (error != hipSuccess) printf("CAN'T FREE ACCUM_BUFFER_DEV IN pathtracer.cu : %s", hipGetErrorString(error));
		error = hipFree(display_buffer_dev);
		if (error != hipSuccess) printf("CAN'T FREE DISPLAY_BUFFER_DEV IN pathtracer.cu : %s", hipGetErrorString(error));
		error = hipFree(pathtracer_buffer_dev);
		if (error != hipSuccess) printf("CAN'T FREE PATHTRACER_BUFFER_DEV IN pathtracer.cu : %s", hipGetErrorString(error));

		hipFree(cudaTriangleList);
		hipFree(cudaBVHNodes);
		hipFree(cudaTrianglesIndex);
		hipFree(cudaMaterialList);

		printf("Everything has been free !\n");
	}
	// device code

	// device constants (like triangles or something)
	

	//pathtracer

	__device__ inline bool triangleIntersect(Hit& hit, Ray ray, Triangle tri, Material mat) {
		float3 edge1 = tri.b - tri.a;
		float3 edge2 = tri.c - tri.a;
		float3 h = cross(ray.d, edge2);
		float a = dot(edge1, h);
		if (a > -0.0001f && a < 0.0001f) return false;
		float f = 1. / a;
		float3 s = ray.o - tri.a;
		float u = f * dot(s, h);
		if (u < 0. || u > 1) return false;
		float3 q = cross(s, edge1);
		float v = f * dot(ray.d, q);
		if (v < 0 || u + v > 1) return false;
		float t = f * dot(edge2, q);

		if (t < 0. || t > hit.t) return false;
		float3 oldAbsorption = hit.mat.extinction;

		hit.t = t;
		hit.mat = mat;
		hit.hit = true;

		float w = 1. - u - v;

		//some shit to refine here
		float3 normalCalc = normalize(cross(edge1, edge2));
		float mult = -sign(dot(ray.d, normalCalc));
		normalCalc *= mult;
		float3 normalFile = normalize(tri.nA * w + tri.nB * u + tri.nC * v) * mult;
		hit.normal = normalFile;

		// it would be smarter to put this inside intersectBVH but i'm lazy now
		float2 samplePos = tri.tA * w + tri.tB * u + tri.tC * v;
		if (mat.useTexture) {
			float4 read = tex2D<float4>(mat.diffuseTexture, samplePos.x, samplePos.y);
			hit.mat.baseColor = make_float3(read.x, read.y, read.z) * read.w;
			hit.mat.alpha = read.w;
		}
		if (mat.use_mapPr) {
			float read = tex2D<float>(mat.roughnessTexture, samplePos.x, samplePos.y);
			hit.mat.roughness = read;
		}
		if (mat.use_mapPm) {
			float read = tex2D<float>(mat.metallicTexture, samplePos.x, samplePos.y);
			hit.mat.metallic = read;
		}
		if (mat.use_mapNor) {
			float3 T, B;
			Onb(hit.normal, T, B);
			float4 read = tex2D<float4>(mat.normalTexture, samplePos.x, samplePos.y);
			hit.normal = ToWorld(T, B, hit.normal, make_float3(read.x * 2.f - 1.f, read.y * 2.f - 1.f, read.z * 2.f - 1.f));
		}
		//hit.normal = dot(hit.normal, ray.d) < 0.f ? hit.normal : normalCalc;
		//hit.normal *= sign(dot(hit.normal, normalCalc));
		hit.normal = normalize(mix(hit.normal, normalCalc, smoothstep(-0.02, 0.1, dot(hit.normal, ray.d))));
		return true;
	}

	__device__ inline bool BVHIntersect(Hit& hit, Ray ray, int3& debug, Triangle* cudaTriList, int* cudaTriIndex, BVH_Node* cudaNodes, Material* cudaMats, bool sunRay) {
		//BVH_Node stack[10];
		int stack[10];
		int stackIdx = 0;
		stack[stackIdx++] = 0;

		float dst = hit.t;
		float3 normal = make_float3(0., 0., 0.);
		bool hasIntersected = false;
		float2 minMax = make_float2(-1, dst);

		float3 invDir = 1. / ray.d;

		while (stackIdx > 0)
		{
			BVH_Node node = cudaNodes[stack[--stackIdx]];
			if (boxIntersectF(hit.t, ray, node.aabbMin, node.aabbMax, invDir) < hit.t) {
				if (node.triCount > 0) { // leaf node
					for (int i = 0; i < node.triCount; i++) { // leaf node
						if (triangleIntersect(hit, ray, cudaTriList[cudaTriIndex[i + node.leftFirst]], cudaMats[cudaTriList[cudaTriIndex[i + node.leftFirst]].matIndex])) {
							debug.z++;
							if (sunRay) return true;
						}
					}
				}
				else {
					BVH_Node childLeft = cudaNodes[node.leftFirst];
					BVH_Node childRight = cudaNodes[node.leftFirst + 1];

					float dstLeft = boxIntersectF(hit.t, ray, childLeft.aabbMin, childLeft.aabbMax, invDir);
					float dstRight = boxIntersectF(hit.t, ray, childRight.aabbMin, childRight.aabbMax, invDir);
					int left = node.leftFirst, right = node.leftFirst + 1;

					if (dstLeft > dstRight) {
						if (dstLeft < hit.t) stack[stackIdx++] = left;
						if (dstRight < hit.t) stack[stackIdx++] = right;
					}
					else {
						if (dstRight < hit.t) stack[stackIdx++] = right;
						if (dstLeft < hit.t) stack[stackIdx++] = left;
					}

				}
				debug.x++;
			}
		}
		return false;
	}
	__device__ inline float3 shadowRay(Hit info, Ray sunRay, Ray ray, Triangle* cudaTriList, int* cudaTriIndex, BVH_Node* cudaNodes, Material* cudaMats, int3& debug) {
		if (BVHIntersect(info, sunRay, debug, cudaTriList, cudaTriIndex, cudaNodes, cudaMats, true)) {
			return make_float3(0.f, 0.f, 0.f);
		}
		float pdf;
		float3 value = Disney::DisneyEval(info, ray.d * 1.0f, info.normal, sunRay.d, pdf);
		return value;
	}
	__device__ inline Hit map(Ray ray, int3 &debug, Triangle* cudaTriList, int* cudaTriIndex, BVH_Node* cudaNodes, Material* cudaMats) {
		Hit hit;
		hit.mat = Material();
		
		//triangleIntersect(hit, ray, Triangle(make_float3(0., 0., 0.), make_float3(3., 0., 0.), make_float3(0., 0., 3.)), Material(make_float3(0.2, 0.3, 0.7), make_float3(1., 1., 1.), 0.02, 0.6, 0.98, make_float3(0., 0., 0.), 0., 1., make_float3(0., 0., 0.), 1.5));

		//boxIntersect(hit, ray, make_float3(-100.0f, 270.0f, -100.0f), make_float3(100.0f, 290.f, 100.0f), Material(make_float3(1., 1., 1.), make_float3(1., 1., 1.), 0.0, 0., 0., make_float3(0.f, 0.f, 0.f), 0.f, 1.f, make_float3(15.f, 15.f, 15.f), make_float3(1.5f, 1.5f, 1.5f)));

		//boxIntersect(hit, ray, make_float3(-400.f, 100.f, 0.f), make_float3(-270.f, 200.f, 230.f), Material(make_float3(1., 1., 1.), make_float3(1., 1., 1.), 0.0, 0., 0., make_float3(0.f, 0.f, 0.f), 0.f, 1.f, make_float3(15.f, 15.f, 15.f), make_float3(1.5f, 1.5f, 1.5f)));
		//boxIntersect(hit, ray, make_float3(-100.0f, 270.0f, -100.0f), make_float3(100.0f, 290.f, 100.0f), Material(make_float3(1., 1., 1.), make_float3(1., 1., 1.), 0.0, 0., 0., make_float3(0.f, 0.f, 0.f), 0.f, 1.f, make_float3(15.f, 15.f, 15.f), make_float3(1.5f, 1.5f, 1.5f)));
		//sphereIntersect(hit, ray, make_float4(18.3, 8., 11.7, 0.75), Material(make_float3(1., 1., 1.), make_float3(1., 1., 1.), 0., 0., 0.9, make_float3(0., 0., 0.), 0., 1., make_float3(10., 2.8, 0.039) * 2., 1.5));
		//sphereIntersect(hit, ray, make_float4(2.6, 15.1, 20.3, 0.75), Material(make_float3(1., 1., 1.), make_float3(1., 1., 1.), 0., 0., 0.9, make_float3(0., 0., 0.), 0., 1., make_float3(10., 2.8, 0.039) * 2., 1.5));
		
		//planeIntersect(hit, ray, make_float3(0., 1., 0.), -2., Material(make_float3(0.1, 0.1, 0.2), make_float3(1., 1., 1.), 0.0, 0., 0., make_float3(0.f, 0.f, 0.f), 0.f, 1.f, make_float3(0.f, 0.f, 0.f), make_float3(1.5f, 1.5f, 1.5f)));
		BVHIntersect(hit, ray, debug, cudaTriList, cudaTriIndex, cudaNodes, cudaMats, 0);
		//__syncthreads();

		/*Material sphMat;
		sphMat.metallic = 0.;
		sphMat.baseColor = make_float3(1., 0.4, 0.6);
		sphMat.roughness = 0.4;
		sphMat.anisotropic = 0.;
		sphereIntersect(hit, ray, make_float4(-400., 50., 0., 50.), sphMat);
		sphMat.anisotropic = 0.2;
		sphereIntersect(hit, ray, make_float4(-250., 50., 0., 50.), sphMat);
		sphMat.anisotropic = 0.4;
		sphereIntersect(hit, ray, make_float4(-100., 50., 0., 50.), sphMat);
		sphMat.anisotropic = 0.6;
		sphereIntersect(hit, ray, make_float4(50., 50., 0., 50.), sphMat);
		sphMat.anisotropic = 0.8;
		sphereIntersect(hit, ray, make_float4(200., 50., 0., 50.), sphMat);
		sphMat.anisotropic = 0.99;
		sphereIntersect(hit, ray, make_float4(350., 50., 0., 50.), sphMat);
		*/
		
		return hit;
	}
	__device__ float HG(float g, float sundotrd) {
		float gg = g * g;	return (1. - gg) / pow(1. + gg - 2. * g * sundotrd, 1.5);
	}
	__device__ float rayleigh(float u) {
		return 0.75 * (1. + clamp(u * u, 0., 1.));
	}
	__device__ float3 skyGradient(float3 rd, kernelParams params) {
		float3 sunColor = mix(make_float3(1., 0.9, 0.8), make_float3(1., 0.2, 0.), exp(-abs(params.sunDirection.y) * 5.));
		float costh0 = dot(rd, params.sunDirection);
		float blend = smoothstep(-1., 0.6, costh0);
		float mult = smoothstep(-0.5, 0.7, costh0) * 0.5 + 0.5;
		float3 day = mix(make_float3(0.3, 0.6, 1.) * 0.7 * rayleigh(costh0), make_float3(0.9, 0.95, 1.), exp(-abs(rd.y) * 5.));
		float3 even = mix(make_float3(0.3, 0.6, 1.) * 0.7, mix(make_float3(0.2, 0.3, 0.4), make_float3(1., 0.2, 0.), blend), exp(-abs(rd.y) * 5.));
		float3 sun = 0.02 / length(rd - params.sunDirection) * sunColor ;

		float3 night = make_float3(0.05, 0.1, 0.2) + clamp(0.02 / length(rd + params.sunDirection), 0., 1.) * make_float3(0.7, 0.8, 1.) * 0.6;
		float3 dayF = mix(day, even, exp(-abs(params.sunDirection.y) * 5.)) + sun;

		float earth = smoothstep(-0.02, 0.02, rd.y) * 0.5 + 0.5;

		float mieM = 1. - abs(dot(params.sunDirection, make_float3(0., 1., 0.))) + 0.3;
		return mix(night, dayF + HG(0.6, costh0) * sunColor * 0.3 * mieM, smoothstep(-0.2, 0.0, params.sunDirection.y)) * earth * 0.75;
		//return vec3(HG(0.9,costh0)) ;
	}
	__device__ inline float3 skyColor(float3 rayDir, kernelParams params) {
		//return make_float3(0.f, 0.f, 0.f);
		//float3 skyGrad = mix(make_float3(0.4, 0.7, 1.), make_float3(0.8, 0.9, 1.), exp(-abs(rayDir.y) * 5.)) * 0.5;
		//return skyGrad * 3.;
		//float sineV = fmax(sin(rayDir.x * 7.) * sin(rayDir.z * 7.), 0.);
		//float sineV = fmaxf(sin(rayDir.x * 18.), 0.f);
		//return make_float3(sineV, sineV, sineV);
		//return make_float3(1., 1., 1.) * fmax((double)rayDir.y, 0.);
		//return dot(rayDir, params.sunDirection) > 0.95 ? make_float3(20., 20., 20.) : make_float3(0.,0.,0.);
		//return skyGrad + smoothstep(1., 0.9, length(rayDir - params.sunDirection) / 0.25) * 50.;
		//return skyGradient(rayDir, params);
		//return make_float3(5.f, 5.f, 5.f);
		const float4 texVal = tex2D<float4>(params.cubeMap,
			atan2f(rayDir.z, rayDir.x) * (float)(0.5f / PI) + 0.5f, 1.f - (rayDir.y * 0.5f + 0.5f));
		return make_float3(texVal.x, texVal.y, texVal.z);
	}
	__device__ inline void pathtrace(float& result, kernelParams params, Ray ray, Rand_state& state, int channel, Triangle* cudaTriList, int* cudaTriIndex, BVH_Node* cudaNodes, Material* cudaMats) {
		float rayColor = 1.0;

		Hit info;
		bool inside = false;
		int3 nullVal = make_int3(0, 0, 0);
		for (int i = 0; i < 50; i++) {
			info = map(ray, nullVal, cudaTriList, cudaTriIndex, cudaNodes, cudaMats);
			info.isInside = inside;

			if (!info.hit) {
				result += getN(skyColor(ray.d, params), channel) * rayColor;
				return;
			}

			ray.o += ray.d * info.t;
			info.mat.roughness = fmaxf(info.mat.roughness, 0.001f);

			if (info.isInside)
				rayColor *= exp(-getN(info.mat.extinction * info.t, channel));
			/*
			// material prop
			float specularChance = 0.;
			float refractionChance = info.mat.refractionProba;

			float rayProba = 1.;
			if (info.mat.f90 > 0.) {
				specularChance = FresnelReflectAmount(info.isInside ? getN(info.mat.IOR, channel) : 1.0f, 
					!info.isInside ? getN(info.mat.IOR, channel) : 1.0f, 
					ray, info);
			}
			//float isSpecular = rand(&state) < specularChance ? 1. : 0.;
			float isSpecular = 0.;
			float isRefractive = 0.;
			float zeta = randC(&state);
			if (specularChance > 0. && zeta < specularChance) {
				isSpecular = 1.;
				rayProba = specularChance;
				ray.o += info.normal * 0.1;
			}
			else if (refractionChance > 0. && zeta < refractionChance + specularChance) {
				isRefractive = 1.;
				rayProba = refractionChance;
				ray.o -= info.normal * 0.1;
			}
			else {
				rayProba = 1. - (specularChance + refractionChance);
				ray.o += info.normal * 0.1;
			}

			float3 diffuseRay = normalize(info.normal + generateUniformSample(state));
			float3 specularRay = reflect(ray.d, info.normal);
			specularRay = normalize(mix(diffuseRay, specularRay, info.mat.specularSmoothness * info.mat.specularSmoothness));

			float3 refractionRayDir = refract(ray.d, info.normal, getN(info.isInside ? info.mat.IOR : 1. / info.mat.IOR, channel));

			ray.d = mix(diffuseRay, specularRay, isSpecular);
			if (dot(refractionRayDir, refractionRayDir) == 0.f) refractionRayDir = ray.d; // should be useless unless f90 == 0
			ray.d = mix(ray.d, refractionRayDir, isRefractive);

			result += getN(info.mat.emissive, channel) * rayColor;

			if (isRefractive < 0.5)
				rayColor *= mix(getN(info.mat.albedo, channel), getN(info.mat.specularAlbedo, channel), isSpecular);
			else inside = !inside;

			rayProba = fmax(rayProba, 0.0001f);
			rayColor /= rayProba;
			*/

			float rC = randC(&state);
			if (rC < info.mat.alpha) {
				float3 L;
				float pdf;
				float3 bsdf = Disney::DisneySample(info, ray.d * -1.f, info.normal, L, pdf, state, ray.o);
				inside = info.isInside;

				ray.d = L;

				result += getN(info.mat.emissive, channel) * rayColor;

				if (pdf > 0.f) rayColor *= getN(bsdf / pdf, channel);
				else return;
			}
			ray.o += ray.d * 0.01f;

			// i sould add MIS on skybox 
			//result += getN(shadowRay(info, Ray(ray.o, params.sunDirection), ray, cudaTriList, cudaTriIndex, cudaNodes, cudaMats, nullVal), channel) * rayColor;
			
			// russian roulette 
			{
				float p = rayColor;
				if (randC(&state) > p)
					return;

				rayColor *= 1.0f / p;
			}
		}
		return;
	}
	__device__ inline float3 pixelColor(kernelParams params, Ray ray, Rand_state& state, Triangle* cudaTriList, int* cudaTriIndex, BVH_Node* cudaNodes, Material* cudaMats) {
		float3 color = make_float3(0., 0., 0.);
		if (!params.isRendering) {
			int3 debugV = make_int3(0, 0, 0);
			Hit info = map(ray, debugV, cudaTriList, cudaTriIndex, cudaNodes, cudaMats);
			color = info.normal * 0.5 + 0.5;
			//float4 test = tex2D<float4>(cudaMats[0].diffuseTexture, 0.f, 0.f);
			//color = dot(ray.d, info.normal) < 0.f ? make_float3(0.f, 0.f, 1.f) : make_float3(1.f, 0.f, 0.f);
			if (dot(info.normal, info.normal) == 0.) color = skyColor(ray.d, params);

			if (info.hit) {
				float pdf;
				color = Disney::DisneyEval(info, ray.d * -1.0f, info.normal, params.sunDirection, pdf);
				if (pdf > 0.f) color /= pdf;

				float3 point = ray.o + ray.d * info.t + info.normal * 0.01;
				color *= shadowRay(info, Ray(point, params.sunDirection), ray, cudaTriList, cudaTriIndex, cudaNodes, cudaMats, debugV);
			} else color = skyColor(ray.d, params);
		}
		else {
			int channel = int(randC(&state) * 3.);
			switch (channel)
			{
			case 0:
				pathtrace(color.x, params, ray, state, 0, cudaTriList, cudaTriIndex, cudaNodes, cudaMats);
				break;
			case 1:
				pathtrace(color.y, params, ray, state, 1, cudaTriList, cudaTriIndex, cudaNodes, cudaMats);
				break;
			case 2:
				pathtrace(color.z, params, ray, state, 2, cudaTriList, cudaTriIndex, cudaNodes, cudaMats);
				break;
			}
			//pathtrace(color.x, params, ray, state, 0);
			//pathtrace(color.y, params, ray, state, 1);
			//pathtrace(color.z, params, ray, state, 2);
			color *= 3.;
		}

		return color;
	}
	__global__ inline void renderPixel(kernelParams params, uint8_t* ptBuffer, float3* accumBuff, Triangle* cudaTriList, int* cudaTriIndex, BVH_Node* cudaNodes, Material* cudaMats) {
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;
		//__syncthread();
		if (x > params.windowSize.x && y > params.windowSize.y) return;

		int idx = x + y * params.windowSize.x;

		Rand_state rand_state;
		hiprand_init(idx, 0, 4096 * params.frameIndex, &rand_state);

		float2 uv = make_float2((float)x / params.windowSize.x, 1. - (float)y / params.windowSize.y);
		float2 uvCam = uv - 0.5;
		uvCam.x *= (float)params.windowSize.x / params.windowSize.y;

		// rayDirection computations
		float fov = 1.;
		float3 cameraTarget = params.rayOrigin + make_float3(sin(params.cameraAngle.x) * cos(params.cameraAngle.y), sin(params.cameraAngle.y), -cos(params.cameraAngle.x) * cos(params.cameraAngle.y));
		float3 ww = normalize(cameraTarget - params.rayOrigin);
		float3 uu = normalize(cross(ww, make_float3(0., 1., 0.)));
		float3 vv = normalize(cross(uu, ww));
		float3 rayDirection = normalize(uu * uvCam.x + vv * uvCam.y + ww * fov);

		/*if (params.isRendering) {
			float3 target = params.rayOrigin + rayDirection * params.focalDistance;
			params.rayOrigin += generateUniformSample(rand_state) * params.DOF_strenght;
			rayDirection = normalize(target - params.rayOrigin);
		}*/
		float3 color0 = max3(pixelColor(params, Ray(params.rayOrigin, rayDirection), rand_state, cudaTriList, cudaTriIndex, cudaNodes, cudaMats), make_float3(0.f,0.f,0.f));

		__syncthreads();
		if (params.frameIndex == 0 || !params.isRendering)
			accumBuff[idx] = color0;
		else
				accumBuff[idx] += color0;

		float3 color = accumBuff[idx] / (params.isRendering ? params.frameIndex + 1 : 1.);
		aces(color);
		// drawing to texture
		int colorX = color.x * 255;
		int colorY = color.y * 255;
		int colorZ = color.z * 255;

		ptBuffer[idx * 4] = uint8_t(clamp(colorX, 0, 255));
		ptBuffer[idx * 4 + 1] = uint8_t(clamp(colorY, 0, 255));
		ptBuffer[idx * 4 + 2] = uint8_t(clamp(colorZ, 0, 255));
		ptBuffer[idx * 4 + 3] = uint8_t(255);
	}
	

	__global__ inline void antialias(kernelParams params, uint8_t* ptBuffer, uint8_t* dispBuffer) {
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;
		//__syncthread();
		if (x > params.windowSize.x && y > params.windowSize.y) return;
		if (x <= 1 || x >= params.windowSize.x - 1 || y <= 1 || y >= params.windowSize.y - 1) return;

		int idx = x + y * params.windowSize.x;
		int left = idx - 1;
		int right = idx + 1;
		int top = idx - params.windowSize.x;
		int bottom = idx + params.windowSize.x;

		int colorR = (int)ptBuffer[idx * 4] +
					 (int)ptBuffer[left * 4] * 0.5 +
					 (int)ptBuffer[right * 4] * 0.5 +
					 (int)ptBuffer[top * 4] * 0.5 +
					 (int)ptBuffer[bottom * 4] * 0.5;

		int colorG = (int)ptBuffer[idx * 4 + 1] +
					 (int)ptBuffer[left * 4 + 1] * 0.5 +
					 (int)ptBuffer[right * 4 + 1] * 0.5 +
					 (int)ptBuffer[top * 4 + 1] * 0.5 +
					 (int)ptBuffer[bottom * 4 + 1] * 0.5;

		int colorB = (int)ptBuffer[idx * 4 + 2] +
					 (int)ptBuffer[left * 4 + 2] * 0.5 +
					 (int)ptBuffer[right * 4 + 2] * 0.5 +
					 (int)ptBuffer[top * 4 + 2] * 0.5 +
					 (int)ptBuffer[bottom * 4 + 2] * 0.5;

		dispBuffer[idx * 4] = uint8_t(colorR * 0.3333f);
		dispBuffer[idx * 4 + 1] = uint8_t(colorG * 0.3333f);
		dispBuffer[idx * 4 + 2] = uint8_t(colorB * 0.3333f);
		dispBuffer[idx * 4 + 3] = uint8_t(255);
	}
	void render(kernelParams params) {
		const int threadSize = 16;
		dim3 blockSize(8, 8, 1U);
		dim3 gridSize(int(params.windowSize.x / blockSize.x), int(params.windowSize.y / blockSize.y), 1U);
		renderPixel<<<gridSize, blockSize >>>(params, display_buffer_dev, accum_buffer_dev, cudaTriangleList, cudaTrianglesIndex, cudaBVHNodes, cudaMaterialList);
		
		//antialias<<<gridSize, blockSize>>>(params, pathtracer_buffer_dev, display_buffer_dev);
		if (hipPeekAtLastError() != hipSuccess) {
			printf("Error with kernel : %s \n", hipGetErrorString(hipGetLastError()));
		}

		hipDeviceSynchronize();
		// copy device display buff. to host
		hipError_t error = hipMemcpy(params.pixelBuffer, display_buffer_dev, params.windowSize.x * params.windowSize.y * 4 * sizeof(uint8_t), hipMemcpyDeviceToHost);
		if (error != hipSuccess) {
			printf("ERROR WHILE TRANSFERING DEVICE DATA TO HOST : %s \n", hipGetErrorString(error));
		}
	}
}